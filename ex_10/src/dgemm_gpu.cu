#include "hip/hip_runtime.h"
extern "C" {
    #include "get_time.h"
}

#include <hipblas.h>
#include <hiprand.h>
#include <stdio.h>
#include <stdlib.h>

#include "get_time.h"

// Fill the array A(nr_rows_A, nr_cols_A) with random numbers on GPU
void GPU_fill_rand(double *A, int nr_rows_A, int nr_cols_A) {
    // Create a pseudo-random number generator
    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

    // Set the seed for the random number generator using the system clock
    hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

    // Fill the array with random numbers on the device
    hiprandGenerateUniformDouble(prng, A, nr_rows_A * nr_cols_A);
}

// Multiply the arrays A and B on GPU and save the result in C
// C(m,n) = A(m,k) * B(k,n)
void gpu_blas_mmul(const double *A, const double *B, double *C, const int m, const int k, const int n) {
    int lda=m,ldb=k,ldc=m;
    const double alf = 1;
    const double bet = 0;
    const double *alpha = &alf;
    const double *beta = &bet;

    // Create a handle for CUBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Do the actual multiplication
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);

    // Destroy the handle
    hipblasDestroy(handle);
}

//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
/*void print_matrix(const double *A, int nr_rows_A, int nr_cols_A) {
    for(int i = 0; i < nr_rows_A; ++i){
        for(int j = 0; j < nr_cols_A; ++j){
            std::cout << A[j * nr_rows_A + i] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}*/

int main(int argc, char *argv[]) {
    // Allocate 3 arrays on CPU
    int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;

    // for simplicity we are going to use square arrays
    nr_rows_A = nr_cols_A = nr_rows_B = nr_cols_B = nr_rows_C = nr_cols_C = strtold(argv[1], NULL);

    double *h_A = (double *)malloc(nr_rows_A * nr_cols_A * sizeof(double));
    double *h_B = (double *)malloc(nr_rows_B * nr_cols_B * sizeof(double));
    double *h_C = (double *)malloc(nr_rows_C * nr_cols_C * sizeof(double));

    // Allocate 3 arrays on GPU
    double *d_A, *d_B, *d_C;
    hipMalloc(&d_A,nr_rows_A * nr_cols_A * sizeof(double));
    hipMalloc(&d_B,nr_rows_B * nr_cols_B * sizeof(double));
    hipMalloc(&d_C,nr_rows_C * nr_cols_C * sizeof(double));

    // Fill the arrays A and B on GPU with random numbers
    GPU_fill_rand(d_A, nr_rows_A, nr_cols_A);
    GPU_fill_rand(d_B, nr_rows_B, nr_cols_B);

    // Optionally we can copy the data back on CPU and print the arrays
    /*hipMemcpy(h_A,d_A,nr_rows_A * nr_cols_A * sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(h_B,d_B,nr_rows_B * nr_cols_B * sizeof(double),hipMemcpyDeviceToHost);
    std::cout << "A =" << std::endl;
    print_matrix(h_A, nr_rows_A, nr_cols_A);
    std::cout << "B =" << std::endl;
    print_matrix(h_B, nr_rows_B, nr_cols_B);*/

    // Multiply A and B on GPU

    uint64_t start = get_time_us();

    gpu_blas_mmul(d_A, d_B, d_C, nr_rows_A, nr_cols_A, nr_cols_B);

    uint64_t runtime = get_time_us() - start;
    printf("floating-point performance: %lf\n", 2.0 * nr_rows_A * nr_rows_A * nr_rows_A / runtime * 1000000);

    // Copy (and print) the result on host memory
    /*hipMemcpy(h_C,d_C,nr_rows_C * nr_cols_C * sizeof(double),hipMemcpyDeviceToHost);
    std::cout << "C =" << std::endl;
    print_matrix(h_C, nr_rows_C, nr_cols_C);*/

    //Free GPU memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free CPU memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
