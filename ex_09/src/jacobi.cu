#include "hip/hip_runtime.h"
extern "C" {
    #include "get_time.h"
}

#include <stdio.h>
#include <stdlib.h>

#define ARRAY_SIZE (1*1024*1024*1024)
#define ARRAY_ELEMENTS (ARRAY_SIZE/8)

#ifndef COPY_TIME
    #define COPY_TIME (0)
#endif

__global__ void triad(double* A, double* B, double* C, double c) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    A[x] = B[x] * c + C[x];
    return;
}

int main(int argc, char *argv[]) {
    uint64_t start = 0, stop = 0, actual_runtime = 0;

    // Allocate and initialize the arrays B and C in the CPU memory and then copy them into
    // the GPU memory
    double* B = (double *)malloc(ARRAY_SIZE);
    double* C = (double *)malloc(ARRAY_SIZE);

    for (int i = 0; i < ARRAY_ELEMENTS; i++) {
        B[i] = i + 0.5;
        C[i] = i + 0.75;
    }

    double* device_B;
    double* device_C;

    hipMalloc((void**)&device_B, ARRAY_SIZE);
    hipMalloc((void**)&device_C, ARRAY_SIZE);

    start = get_time_us();
    hipMemcpy(device_B, B, ARRAY_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(device_C, C, ARRAY_SIZE, hipMemcpyHostToDevice);
    stop = get_time_us();
    #if COPY_TIME == 1
        actual_runtime += stop - start;
    #endif

    double* A = (double *)malloc(ARRAY_SIZE);
    double* device_A;
    hipMalloc((void**)&device_A, ARRAY_SIZE);

    // Call your kernel function to run the STREAM Triad on the GPU
    start = get_time_us();
    triad<<<ARRAY_ELEMENTS/1024,1024>>>(device_A, device_B, device_C, 0.5);
    hipDeviceSynchronize();
    stop = get_time_us();
    actual_runtime += stop - start;

    // Finally, copy array A from the device memory back to the host memory to verify the
    // correctness of your kernel implementation
    double* verify = (double *)malloc(ARRAY_SIZE);
    for (int i = 0; i < ARRAY_ELEMENTS; ++i) {
        verify[i] = B[i] * 0.5 + C[i];
    }

    start = get_time_us();
    hipMemcpy(A, device_A, ARRAY_SIZE, hipMemcpyDeviceToHost);
    stop = get_time_us();
    #if COPY_TIME == 1
        actual_runtime += stop - start;
    #endif

    for (int i = 0; i < ARRAY_ELEMENTS; ++i) {
        if (verify[i] != A[i]) {
            printf("verify: failed\n");
            return 1;
        }
    }

    hipFree(device_A);
    hipFree(device_B);
    hipFree(device_C);
    free(A);
    free(B);
    free(C);
    free(verify);

    double bandwidth = 3.0 * ARRAY_SIZE / actual_runtime;
    printf("bandwidth: %lf\n", bandwidth);

    return 0;
}
