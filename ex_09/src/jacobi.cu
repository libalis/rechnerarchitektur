#include "hip/hip_runtime.h"
extern "C" {
    #include "get_time.h"
}

#include <stdio.h>
#include <stdlib.h>

#ifndef COPY_TIME
    #define COPY_TIME (0)
#endif

#ifndef MIN_RUNTIME
    #define MIN_RUNTIME (100)
#endif

__global__ void update_grid(double* grid_source, double* grid_target, uint32_t dx, uint32_t dy) {
    // Switch the pointers for next iteration
	double* tmp = grid_source;
	grid_source = grid_target;
	grid_target = tmp;

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    x++;
    y++;

    grid_target[y * dx + x] = grid_source[(y - 1) * dx + x] + grid_source[y * dx + (x - 1)];
    grid_target[y * dx + x] += grid_source[y * dx + (x + 1)] + grid_source[(y + 1) * dx + x];
    grid_target[y * dx + x] *= 0.25;

    return;
}

int main(int argc, char *argv[]) {
    uint64_t dx = strtold(argv[1], NULL);
	uint64_t dy = strtold(argv[2], NULL);

    uint64_t start = 0, stop = 0, actual_runtime = 0, runs = 0, runtime = 0;

    // Allocate and initialize the arrays B and C in the CPU memory and then copy them into
    // the GPU memory
    double* grid_source = (double *)malloc(dx * dy * sizeof(double));
    double* grid_target = (double *)malloc(dx * dy * sizeof(double));

    for (int y = 0; y < dy; y++) {
		for (int x = 0; x < dx; x++) {
			if (y == 0 || x == 0) {
				grid_source[y * dx + x] = 1.0;
				grid_target[y * dx + x] = 1.0;
			} else {
				grid_source[y * dx + x] = 0.0;
				grid_target[y * dx + x] = 0.0;
			}
		}
	}

    double* device_grid_source;
    double* device_grid_target;

    hipMalloc((void**)&device_grid_source, dx * dy * sizeof(double));
    hipMalloc((void**)&device_grid_target, dx * dy * sizeof(double));

    start = get_time_us();
    hipMemcpy(device_grid_source, grid_source, dx * dy * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_grid_target, grid_target, dx * dy * sizeof(double), hipMemcpyHostToDevice);
    stop = get_time_us();
    #if COPY_TIME == 1
        actual_runtime += stop - start;
    #endif

    // Measure for MIN_RUNTIME with 100 ms, 1s and 10s
    for (runs = 1u; runtime < MIN_RUNTIME; runs = runs << 1u) {
        // Call your kernel function to run jacobi on the GPU
        start = get_time_us();
        update_grid<<<(dx - 2) * (dy - 2) / 1024, 1024>>>(device_grid_source, device_grid_target, dx, dy);
        hipDeviceSynchronize();
        stop = get_time_us();
        runtime = stop - start;
    }

    actual_runtime += runtime;

    // Finally, copy array A from the device memory back to the host memory to verify the
    // correctness of your kernel implementation
    double* verify = (double *)malloc(dx * dy * sizeof(double));
    for (runs = 1u; runtime < MIN_RUNTIME; runs = runs << 1u) {
        start = get_time_us();
        for (int y = 1; y < dy - 1; y++) {
            for (int x = 1; x < dx - 1; x++) {
                // Switch the pointers for next iteration
                double* tmp = grid_source;
                grid_source = verify;
                verify = tmp;

                verify[y * dx + x] = grid_source[(y - 1) * dx + x] + grid_source[y * dx + (x - 1)];
                verify[y * dx + x] += grid_source[y * dx + (x + 1)] + grid_source[(y + 1) * dx + x];
                verify[y * dx + x] *= 0.25;
            }
        }
        stop = get_time_us();
        runtime = stop - start;
    }

    start = get_time_us();
    hipMemcpy(grid_target, device_grid_target, dx * dy * sizeof(double), hipMemcpyDeviceToHost);
    stop = get_time_us();
    #if COPY_TIME == 1
        actual_runtime += stop - start;
    #endif

    for (int y = 1; y < dy - 1; y++) {
		for (int x = 1; x < dx - 1; x++) {
            if (verify[y * dx + x] != grid_target[y * dx + x]) {
                printf("verify: failed\n");
                return 1;
            }
        }
    }

    hipFree(device_grid_source);
    hipFree(device_grid_target);
    free(grid_source);
    free(grid_target);
    free(verify);

    double bandwidth = 3.0 * dx * dy * sizeof(double) / actual_runtime;
    printf("%d,%lf\n", MIN_RUNTIME, bandwidth);

    return 0;
}
