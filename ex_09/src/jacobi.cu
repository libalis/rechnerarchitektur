#include "hip/hip_runtime.h"
extern "C" {
    #include "get_time.h"
}

#include <stdio.h>
#include <stdlib.h>

#ifndef COPY_TIME
    #define COPY_TIME (0)
#endif

#ifndef MIN_RUNTIME
    #define MIN_RUNTIME (100)
#endif

__global__ void update_grid(double* grid_source, double* grid_target, uint32_t dx, uint32_t dy) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x == 0 || x >= dx - 1 || y == 0 || y >= dy - 1)
        return;

    grid_target[y * dx + x] = grid_source[(y - 1) * dx + x] + grid_source[y * dx + (x - 1)];
    grid_target[y * dx + x] += grid_source[y * dx + (x + 1)] + grid_source[(y + 1) * dx + x];
    grid_target[y * dx + x] *= 0.25;

    return;
}

int main(int argc, char *argv[]) {
    uint64_t dx = strtold(argv[1], NULL);
    uint64_t dy = strtold(argv[2], NULL);

    uint64_t start = 0, stop = 0, actual_runtime = 0, runs = 0, runtime = 0;

    // Allocate and initialize the arrays B and C in the CPU memory and then copy them into
    // the GPU memory
    double* grid_source = (double *)malloc(dx * dy * sizeof(double));
    double* grid_target = (double *)malloc(dx * dy * sizeof(double));

    for (int y = 0; y < dy; y++) {
        for (int x = 0; x < dx; x++) {
            if (y == 0 || x == 0) {
                grid_source[y * dx + x] = 1.0;
                grid_target[y * dx + x] = 1.0;
            } else {
                grid_source[y * dx + x] = 0.0;
                grid_target[y * dx + x] = 0.0;
            }
        }
    }

    double* device_grid_source;
    double* device_grid_target;

    hipMalloc((void**)&device_grid_source, dx * dy * sizeof(double));
    hipMalloc((void**)&device_grid_target, dx * dy * sizeof(double));

    start = get_time_us();
    hipMemcpy(device_grid_source, grid_source, dx * dy * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_grid_target, grid_target, dx * dy * sizeof(double), hipMemcpyHostToDevice);
    stop = get_time_us();
    #if COPY_TIME == 1
        actual_runtime += stop - start;
    #endif

    uint64_t t = ((dx * dy) % 1024) ? (dx * dy / 1024 + 1) : (dx * dy / 1024);
    uint64_t r = 0u;

    // Measure for MIN_RUNTIME with 100 ms, 1s and 10s
    for (runs = 1u; runtime < MIN_RUNTIME; runs = runs << 1u) {
        // Call your kernel function to run jacobi on the GPU
        start = get_time_us();

        // Switch the pointers for next iteration
        double* tmp = device_grid_source;
        device_grid_source = device_grid_target;
        device_grid_target = tmp;

        update_grid<<<t, 1024>>>(device_grid_source, device_grid_target, dx, dy);
        hipDeviceSynchronize();

        stop = get_time_us();
        runtime = stop - start;

        r++;
    }

    actual_runtime += runtime;

    // Finally, copy array A from the device memory back to the host memory to verify the
    // correctness of your kernel implementation
    double* verify = (double *)malloc(dx * dy * sizeof(double));
    for (runs = 0u; runs < r; runs++) {
        start = get_time_us();
        for (int y = 1; y < dy - 1; y++) {
            for (int x = 1; x < dx - 1; x++) {
                // Switch the pointers for next iteration
                double* tmp = grid_source;
                grid_source = verify;
                verify = tmp;

                verify[y * dx + x] = grid_source[(y - 1) * dx + x] + grid_source[y * dx + (x - 1)];
                verify[y * dx + x] += grid_source[y * dx + (x + 1)] + grid_source[(y + 1) * dx + x];
                verify[y * dx + x] *= 0.25;
            }
        }
        stop = get_time_us();
        runtime = stop - start;
    }

    start = get_time_us();
    hipMemcpy(grid_target, device_grid_target, dx * dy * sizeof(double), hipMemcpyDeviceToHost);
    stop = get_time_us();
    #if COPY_TIME == 1
        actual_runtime += stop - start;
    #endif

    for (int y = 1; y < dy - 1; y++) {
        for (int x = 1; x < dx - 1; x++) {
            if (verify[y * dx + x] - grid_target[y * dx + x] < -0.5 && verify[y * dx + x] - grid_target[y * dx + x] > 0.5) {
                printf("verify: failed\n");
                return 1;
            }
        }
    }

    hipFree(device_grid_source);
    hipFree(device_grid_target);
    free(grid_source);
    free(grid_target);
    free(verify);

    double bandwidth = 2.0 * dx * dy * sizeof(double) / actual_runtime;
    printf("%d,%lf\n", MIN_RUNTIME, bandwidth);

    return 0;
}
