#include "hip/hip_runtime.h"
#include "get_time.h"
#include <stdio.h>
#include <stdlib.h>

#define ARRAY_SIZE (1*1024*1024*1024)
#define ARRAY_ELEMENTS (ARRAY_SIZE/8)

#ifndef COPY_TIME
    #define COPY_TIME (0)
#endif

__global__ uint64_t triad(double* A, double* B, double* C, double c, int n, uint64_t actual_runtime) {
    uint64_t start = get_time_us();

    for (int i = 0; i < n; ++i) {
        A[i] = B[i] * c + C[i];
    }

    // Measure solely the kernel execution time and calculate the bandwidth
    uint64_t stop = get_time_us();
    actual_runtime += stop - start;
    #if COPY_TIME == 0
        double bandwidth = 3.0 * ARRAY_SIZE / actual_runtime;
        printf("bandwidth: %lf\n", bandwidth);
    #endif

    return actual_runtime;
}

int main(int argc, char *argv[]) {
    uint64_t actual_runtime = 0;

    // Allocate and initialize the arrays B and C in the CPU memory and then copy them into
    // the GPU memory
    double* B = malloc(ARRAY_SIZE);
    double* C = malloc(ARRAY_SIZE);

    for (int i = 0; i < ARRAY_ELEMENTS; i++) {
        B[i] = i + 0.5;
        C[i] = i + 0.75;
    }

    double* device_B;
    double* device_C;

    hipMalloc((void**)&device_B, ARRAY_SIZE);
    hipMalloc((void**)&device_C, ARRAY_SIZE);

    uint64_t start = get_time_us();
    hipMemcpy(device_B, B, ARRAY_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(device_C, C, ARRAY_SIZE, hipMemcpyHostToDevice);
    uint64_t stop = get_time_us();
    #if COPY_TIME == 1
        actual_runtime += stop - start;
    #endif

    double* A = malloc(ARRAY_SIZE);
    double* device_A;
    hipMalloc((void**)&device_A, ARRAY_SIZE);

    // Call your kernel function to run the STREAM Triad on the GPU
    // TODO: <<<blocks, threadsPerBlock>>>
    actual_runtime = triad<<<1,1>>>(device_A, device_B, device_C, 0.5, ARRAY_ELEMENTS, actual_runtime);

    hipDeviceSynchronize();

    // Finally, copy array A from the device memory back to the host memory to verify the
    // correctness of your kernel implementation
    double* verify = _mm_malloc(ARRAY_SIZE, 64);
    for (int i = 0; i < ARRAY_ELEMENTS; ++i) {
        verify[i] = B[i] * 0.5 + C[i];
    }

    start = get_time_us();
    hipMemcpy(A, device_A, ARRAY_SIZE, hipMemcpyDeviceToHost);
    stop = get_time_us();
    #if COPY_TIME == 1
        actual_runtime += stop - start;
        double bandwidth = 3.0 * ARRAY_SIZE / actual_runtime;
        printf("bandwidth: %lf\n", bandwidth);
    #endif

    for (int i = 0; i < ARRAY_ELEMENTS; ++i) {
        if (verify[i] != A[i]) {
            printf("verify: failed\n");
            return 1;
        }
    }

    free(A);
    free(B);
    free(C);
    hipFree(device_A);
    hipFree(device_B);
    hipFree(device_C);

    return 0;
}
