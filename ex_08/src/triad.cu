#include "hip/hip_runtime.h"
#include "get_time.h"
#include <stdio.h>
#include <stdlib.h>

#define ARRAY_SIZE (1*1024*1024*1024)
#define ARRAY_ELEMENTS (ARRAY_SIZE/8)

#ifndef COPY_TIME
    #define COPY_TIME (0)
#endif

__global__ void triad(double* A, double* B, double* C, double c, int n, uint64_t actual_runtime) {
    uint64_t start = get_time_us();

    for (int i = 0; i < n; ++i) {
        A[i] = B[i] * c + C[i];
    }

    // Measure solely the kernel execution time and calculate the bandwidth
    uint64_t stop = get_time_us();
    actual_runtime += stop - start;
    double bandwidth = 3.0 * ARRAY_SIZE / actual_runtime;
    printf("bandwidth: %lf\n", bandwidth);
}

int main(int argc, char *argv[]) {
    uint64_t actual_runtime = 0;

    // Allocate and initialize the arrays B and C in the CPU memory and then copy them into
    // the GPU memory
    double* B = _mm_malloc(ARRAY_SIZE, 64);
    double* C = _mm_malloc(ARRAY_SIZE, 64);

    for (int i = 0; i < ARRAY_ELEMENTS; i++) {
        B[i] = i + 0.5;
        C[i] = i + 0.75;
    }

    double* device_B;
    double* device_C;

    hipMalloc((void**)&device_B, ARRAY_SIZE);
    hipMalloc((void**)&device_C, ARRAY_SIZE);

    uint64_t start = get_time_us();
    hipMemcpy(device_B, B, ARRAY_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(device_C, C, ARRAY_SIZE, hipMemcpyHostToDevice);
    uint64_t stop = get_time_us();
    #if COPY_TIME == 1
        actual_runtime += stop - start;
    #endif

    double* A = _mm_malloc(ARRAY_SIZE, 64);
    double* device_A;
    hipMalloc((void**)&device_A, ARRAY_SIZE);

    // Call your kernel function to run the STREAM Triad on the GPU
    triad<<<1,1>>>(device_A, device_B, device_C, 0.5, ARRAY_ELEMENTS, actual_runtime);

    hipDeviceSynchronize();

    // Finally, copy array A from the device memory back to the host memory to verify the
    // correctness of your kernel implementation
    double* verify = _mm_malloc(ARRAY_SIZE, 64);
    for (int i = 0; i < ARRAY_ELEMENTS; ++i) {
        verify[i] = B[i] * 0.5 + C[i];
    }

    start = get_time_us();
    hipMemcpy(A, device_A, ARRAY_SIZE, hipMemcpyDeviceToHost);
    stop = get_time_us();
    #if COPY_TIME == 1
        actual_runtime += stop - start;
    #endif

    for (int i = 0; i < ARRAY_ELEMENTS; ++i) {
        if (verify[i] != A[i]) {
            printf("verify: failed\n");
            return 1;
        }
    }

    _mm_free(A);
    _mm_free(B);
    _mm_free(C);
    hipFree(device_A);
    hipFree(device_B);
    hipFree(device_C);

    return 0;
}
